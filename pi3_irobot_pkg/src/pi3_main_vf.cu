#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <lwpr.h>
#include <math.h>
#include <lwpr_xml.h>
#include "ros/ros.h"
#include "pi3_irobot_pkg/rl_vel.h"
#include "pi3_irobot_pkg/irobot_state.h"

#define CONTROL_DIM 2
#define STATE_DIM 5
#define DERIV_STATE_DIM 3
//N is the number of states in an LWPR model, in this case 5
#define N 5
#define K 100
#define M 16
#define T 60
#define HZ 20
#define MAX_VAR 10.0

#define BLOCKSIZE 1024

//Defines a class for maintaining callbacks updating the current state
class StateUpdater
{
private:
  ros::NodeHandle n;
  ros::Subscriber sub;
public:
  float s[STATE_DIM];
  StateUpdater(float* init_state);
  void init_subscriber();
  void stateCallback(const pi3_irobot_pkg::irobot_state::ConstPtr& state_msg);
};

StateUpdater::StateUpdater(float* init_state) {
  int i;
  for (i = 0; i < STATE_DIM; i++) {
    s[i] = init_state[i];
  }
}

void StateUpdater::stateCallback(const pi3_irobot_pkg::irobot_state::ConstPtr& state_msg) {
  s[0] = state_msg->x;
  s[1] = state_msg->y;
  s[2] = state_msg->theta;
  s[3] = state_msg->r_vel;
  s[3] = state_msg->l_vel;
}

void StateUpdater::init_subscriber() {
  sub = n.subscribe("state", 1, &StateUpdater::stateCallback, this);
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Define a data structure which contains the elements
//of an LWPR receptive field needed to make a prediction.
typedef struct {
  float c[N];
  float D[N*N];
  int trustworthy;
  float beta0;
  float mean_x[N];
  int nReg;
  float n_data[N];
  float U[N*N];
  float P[N*N];
  float beta[N];
  float SSs2[N];
  float sum_e_cv2[N];
  float sum_W[N];
  float SSp;
} RF_Predict;

//Transfers data from a full receptive field to a (smaller) rfPredict struct
void rfTransfer(LWPR_ReceptiveField *rf_orig, RF_Predict *rf_pred, int nInS) {
  int i,j;
  int R = rf_orig->nReg;
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++){
      rf_pred->D[i*N + j] = float(rf_orig->D[nInS*i + j]);
    }
  }
  for (i = 0; i < N; i++) {
    for (j = 0; j < N; j++) {
      if (i < R) {
	rf_pred->U[i*N + j] = float(rf_orig->U[i*nInS + j]);
	rf_pred->P[i*N + j] = float(rf_orig->P[i*nInS + j]);
      }
      else {
	//Pad un-used part of the array with zeros to prevent memory leaks
	rf_pred->U[i*N + j] = 0;
	rf_pred->P[i*N + j] = 0;
      }
    }
  }
  for (i = 0; i < N; i++) {
    rf_pred->c[i] = float(rf_orig->c[i]);
    rf_pred->mean_x[i] = float(rf_orig->mean_x[i]);
  }
  for (i = 0; i < R; i++) {
    rf_pred->n_data[i] = float(rf_orig->n_data[i]);
    rf_pred->beta[i] = float(rf_orig->beta[i]);
    rf_pred->SSs2[i] = float(rf_orig->SSs2[i]);
    rf_pred->sum_e_cv2[i] = float(rf_orig->sum_e_cv2[i]);
    rf_pred->sum_W[i] = float(rf_orig->sum_w[i]);
  }
  for (i = R; i < N; i++) {
    rf_pred->n_data[i] = 0;
    rf_pred->beta[i] = 0;
    rf_pred->SSs2[i] = 0;
    rf_pred->sum_e_cv2[i] = 0;
    rf_pred->sum_W[i] = 0;
  }
  rf_pred->trustworthy = rf_orig->trustworthy;
  rf_pred->beta0 = float(rf_orig->beta0);
  rf_pred->nReg = rf_orig->nReg;
  rf_pred->SSp = float(rf_orig->SSp);
}

//==============================================================================
//----------------------------CUDA FUNCTIONS------------------------------------
//==============================================================================

__constant__ float U_d[T*CONTROL_DIM];
__constant__ float dm_d[T*M*DERIV_STATE_DIM];
__constant__ float norm_in_d[N];

__device__ void print_vec(float* A, float* B, int n) {
  printf("\n\n++++++++++++++++++++++++++++++++++++++++++");
  printf("\n Printing A \n");
  for (int i = 0; i < n; i++) {
    printf("  %f  ", A[i]);
  }
  printf("- \n \n -");
  printf("\n Printing B \n");
  for (int i = 0; i < n; i++) {
    printf("  %f  ", B[i]);
  }
  printf("=================================\n");
}

__device__ void rf_to_shared_mem(RF_Predict *rf_s, RF_Predict *rf_g, int idx) {
  //Smaller indices load arrays
  if (idx < N*N) {
    rf_s->D[idx] = rf_g->D[idx];
  }
  else if (idx >= N*N && idx < 2*N*N) {
    rf_s->U[idx-N*N] = rf_g->U[idx-N*N];
  }
  else if (idx >= 2*N*N && idx < 3*N*N) {
    rf_s->P[idx-2*N*N] = rf_g->P[idx-2*N*N];
  }
  //Intermediate indices load vectors
  else if (idx >= 3*N*N && idx < 3*N*N + N) {
    rf_s->c[idx-3*N*N] = rf_g->c[idx-3*N*N];
  }
  else if (idx >= 3*N*N + N && idx < 3*N*N + 2*N) {
    rf_s->mean_x[idx-(3*N*N + N)] = rf_g->mean_x[idx-(3*N*N + N)];
  }
  else if (idx >= 3*N*N + 2*N && idx < 3*N*N + 3*N) {
    rf_s->n_data[idx-(3*N*N + 2*N)] = rf_g->n_data[idx-(3*N*N + 2*N)];
  }
  else if (idx >= 3*N*N + 3*N && idx < 3*N*N + 4*N) {
    rf_s->beta[idx-(3*N*N + 3*N)] = rf_g->beta[idx-(3*N*N + 3*N)];
  }
  else if (idx >= 3*N*N + 4*N && idx < 3*N*N + 5*N) {
    rf_s->SSs2[idx-(3*N*N + 4*N)] = rf_g->SSs2[idx-(3*N*N + 4*N)];
  }
  else if (idx >= 3*N*N + 5*N && idx < 3*N*N + 6*N) {
    rf_s->sum_e_cv2[idx-(3*N*N + 5*N)] = rf_g->sum_e_cv2[idx-(3*N*N + 5*N)];
  }
  else if (idx >= 3*N*N + 6*N && idx < 3*N*N + 7*N) {
    rf_s->sum_W[idx-(3*N*N + 6*N)] = rf_g->sum_W[idx-(3*N*N + 6*N)];
  }
  //Big indices load scalars
  else if (idx == 3*N*N + 7*N) {
    rf_s->trustworthy = rf_g->trustworthy;
  }
  else if (idx == 3*N*N + 7*N + 1) {
    rf_s->beta0 = rf_g->beta0;
  }
  else if (idx == 3*N*N + 7*N + 2) {
    rf_s->nReg = rf_g->nReg;
  }
  else if (idx == 3*N*N + 7*N + 3) {
    rf_s->SSp = rf_g->SSp;
  }
}

__device__ void compute_proj(int nR, float* s, float* xc, float* U, float* P) {
  int i,j;
  float dot;
  float xu[N];
  for (i = 0; i < N; i++) {
    xu[i] = xc[i];
  }
  for (i = 0; i < nR - 1; i++) {
    dot = 0;
    for (j = 0; j < N; j++) {
      dot += U[i*N + j]*xu[j];
    }
    s[i] = dot;
    for (j = 0; j < N; j++) {
      xu[j] -= s[i]*P[i*N + j];
    }
  }  
  dot = 0;
  for (i = 0; i < N; i++) {
    dot += U[(nR - 1)*N + i]*xu[i];
  }
  s[nR - 1] = dot;
}

__device__ void rf_predict(RF_Predict *rf, float* pred_helper, float* x, int index, int t) {
  int i,j;
  float xc[N];
  for (i = 0; i < N; i++) {
    xc[i] = x[i] - rf->c[i];
  }
  float dist = 0;
  for (i = 0; i < N; i++) {
    float dot = 0;
    for (j = 0; j < N; j++) {
      dot += rf->D[j*N + i]*xc[j];
    }
    dist += xc[i]*dot;
  }
  float w = __expf(-.5*dist);
  float yp_n;
  float sigma2;
  if (w > .001 && rf->trustworthy) {
    yp_n = rf->beta0;
    sigma2 = 0.0;
    for (i = 0; i < N; i++) {
      xc[i] = x[i] - rf->mean_x[i];
    }
    int nR = rf->nReg;
    if (rf->n_data[nR-1] <= 2*N) {
      nR--;
    }
    float s[N];
    compute_proj(nR, s, xc, rf->U, rf->P);
    for (i = 0; i < nR; i++) {
      yp_n += s[i]*rf->beta[i];
      sigma2 += s[i]*s[i] / rf->SSs2[i];
    }
    sigma2 = rf->sum_e_cv2[nR-1]/(rf->sum_W[nR-1] - rf->SSp)*(1+w*sigma2);
    pred_helper[0] = yp_n*w;
    pred_helper[1] = w;
    pred_helper[2] = w*yp_n*yp_n;
    pred_helper[3] = w*sigma2;
  }
  else {
      pred_helper[0] = 0;
      pred_helper[1] = 0;
      pred_helper[2] = 0;
      pred_helper[3] = 0;
  }
}

__device__ void compute_predict_conf(RF_Predict* rfs, float* x, int numRFS, float* vals, int t) {
  int i;
  float pred_helper[] = {0,0,0,0};
  float sum_wy = 0;
  float sum_w = 0;
  float sum_wyy = 0;
  float sum_conf = 0;
  __shared__ RF_Predict rf_s0;
  __shared__ RF_Predict rf_s1;
  __shared__ RF_Predict rf_s2;
  __shared__ RF_Predict rf_s3;
  __shared__ RF_Predict rf_s4;
  __shared__ RF_Predict rf_s5;
  __shared__ RF_Predict rf_s6;
  __shared__ RF_Predict rf_s7;
  int tot_el = 3*N*N + 7*N + 4;
  int idx = threadIdx.x*M + threadIdx.y;
  for (i = 0; i < numRFS; i+= 7) {    
    __syncthreads();
    if (idx < tot_el && i < numRFS) {
      rf_to_shared_mem(&rf_s0, &rfs[i], idx);
    }
    else if (idx >= tot_el && idx < 2*tot_el && i + 1 < numRFS) {
      rf_to_shared_mem(&rf_s1, &rfs[i+1], idx - tot_el);
    }
    else if (idx >= 2*tot_el && idx < 3*tot_el && i + 2 < numRFS) {
      rf_to_shared_mem(&rf_s2, &rfs[i+2], idx - 2*tot_el);
    }
    else if (idx >= 3*tot_el && idx < 4*tot_el && i + 3 < numRFS) {
      rf_to_shared_mem(&rf_s3, &rfs[i+3], idx - 3*tot_el);
    }
    else if (idx >= 4*tot_el && idx < 5*tot_el && i + 4 < numRFS) {
      rf_to_shared_mem(&rf_s4, &rfs[i+4], idx - 4*tot_el);
    }
    else if (idx >= 5*tot_el && idx < 6*tot_el && i + 5 < numRFS) {
      rf_to_shared_mem(&rf_s5, &rfs[i+5], idx - 5*tot_el);
    }
    else if (idx >= 6*tot_el && idx < 7*tot_el && i + 6 < numRFS) {
      rf_to_shared_mem(&rf_s6, &rfs[i+6], idx - 6*tot_el);
    }
    else if (idx >= 7*tot_el && idx < 8*tot_el && i + 7 < numRFS) {
      rf_to_shared_mem(&rf_s7, &rfs[i+7], idx - 7*tot_el);
    }
    __syncthreads();
    rf_predict(&rf_s0, pred_helper, x, i, t);
    sum_wy += pred_helper[0];
    sum_w += pred_helper[1];
    sum_wyy += pred_helper[2];
    sum_conf += pred_helper[3];

    if (i + 1 < numRFS) {
      rf_predict(&rf_s1, pred_helper, x, i+1, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 2 < numRFS) {
      rf_predict(&rf_s2, pred_helper, x, i+2, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 3 < numRFS) {
      rf_predict(&rf_s3, pred_helper, x, i+3, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 4 < numRFS) {
      rf_predict(&rf_s4, pred_helper, x, i+4, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 5 < numRFS) {
      rf_predict(&rf_s5, pred_helper, x, i+5, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 6 < numRFS) {
      rf_predict(&rf_s6, pred_helper, x, i+6, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
    }
    if (i + 7 < numRFS) {
      rf_predict(&rf_s7, pred_helper, x, i+7, t);
      sum_wy += pred_helper[0];
      sum_w += pred_helper[1];
      sum_wyy += pred_helper[2];
      sum_conf += pred_helper[3];
      }
  }
  if (sum_w > 0) {
    vals[0] = sum_wy/sum_w;
    vals[1] = fmin((float)sqrt(fabs(sum_conf + sum_wyy - sum_wy*vals[0]))/sum_w, (float)MAX_VAR);
  }
  else {
    vals[0] = 0;
    vals[1] = MAX_VAR;
  }
}

//Enforces constraints for the robot
__device__ void enforce_constraints(float* s) {
  if (s[0] > 10.0) {
    s[0] = 10.0;
  }
  else if (s[0] < -10.0) {
    s[0] = -10.0;
  }
  if (s[1] > 10.0) {
    s[1] = 10.0;
  }
  else if (s[1] < -10.0) {
    s[1] = -10.0;
  }
  if (s[2] > 3.14) {
    s[2] = -3.14;
  }
  else if (s[2] < -3.14) {
    s[2] = 3.14;
  }
  if (s[3] > .5) {
    s[3] = .5;
  }
  else if (s[3] < -.5) {
    s[3] = -.5;
  }
  if (s[4] > .5) {
    s[4] = .5;
  }
  else if (s[4] < -.5) {
    s[4] = -.5;
  }
}

__device__ void compute_dynamics(float* s, float* u, float* lwpr_input, RF_Predict* rfs1, RF_Predict* rfs2, 
				 RF_Predict* rfs3, float* sigmas, int timestep, int numRFS1, int numRFS2, int numRFS3) 
{
  float dt = 1.0/(1.0*HZ);
  //------Problem Specific------------
  float vals[2];
  //Normalize according to norm_in_d, note that all lwpr models 
  //have the same input, hence the same norm_in, and same input.
  lwpr_input[0] = s[0]/norm_in_d[0];
  lwpr_input[1] = s[1]/norm_in_d[1];
  lwpr_input[2] = s[2]/norm_in_d[2];
  lwpr_input[3] = s[3]/norm_in_d[3];
  lwpr_input[4] = s[4]/norm_in_d[4];
  //Compute the first prediction
  compute_predict_conf(rfs1, lwpr_input, numRFS1, vals, timestep);
  s[0] += dt*(vals[0] + vals[1]*dm_d[T*DERIV_STATE_DIM*threadIdx.y + DERIV_STATE_DIM*timestep]);
  sigmas[0] = vals[1];
  //Compute second prediction
  compute_predict_conf(rfs2, lwpr_input, numRFS2, vals, timestep);
  s[1] += dt*(vals[0] + vals[1]*dm_d[T*DERIV_STATE_DIM*threadIdx.y + DERIV_STATE_DIM*timestep + 1]);
  sigmas[1] = vals[1];
  //Compute third prediction
  compute_predict_conf(rfs3, lwpr_input, numRFS3, vals, timestep);
  s[2] += dt*(vals[0] + vals[1]*dm_d[T*DERIV_STATE_DIM*threadIdx.y + DERIV_STATE_DIM*timestep + 2]);
  sigmas[2] = vals[1];
  //Low pass filter controls
  s[3] += dt*((u[0] + u[1]) - s[3]);
  s[4] += dt*((u[0] - u[1]) - s[4]);
  //Make sure all constraints are satisfied
  enforce_constraints(s);
}

//Computes the immediate cost according to the PI^2 framework.
//TODO: Add control cost and anti-biasing term.
__device__ float compute_cost(float* s, float* u, float* goal, float* sigmas)
{
  float d1 = (s[0] - goal[0]);
  float d2 = (s[1] - goal[1]);
  float cost = d1*d1 + d2*d2;
  return cost;
}

__global__ void rollout_kernel(float* aug_state_costs_d, float* state_d, float* goal_d, RF_Predict* rfs1,
			       RF_Predict* rfs2, RF_Predict* rfs3, float* du_d, float* vars_d, 
			       int numRFS1, int numRFS2, int numRFS3)
{
  int tdx = threadIdx.x;
  int tdy = threadIdx.y;
  int bdx = blockIdx.x;
  if (blockDim.x*bdx+tdx < K) {
    //Initialize the local state
    float s[STATE_DIM];
    float u[CONTROL_DIM];
    float lwpr_input[N];
    float vars[CONTROL_DIM];
    float sigmas[DERIV_STATE_DIM];
    int i,j;
    //Load the initial state
    for (i = 0; i < STATE_DIM; i++) {
      s[i] = state_d[i];
    }
    //Load vars
    for (i = 0; i < CONTROL_DIM; i++) {
      vars[i] = vars_d[i];
    }
    for (i = 0; i < T; i++) {
      //Start the main program loop
      for (j = 0; j < CONTROL_DIM; j++) {
	if (bdx == 0 && tdx == 0) {
	  u[j] = U_d[i*CONTROL_DIM + j];
	}
	else {
	  u[j] = U_d[i*CONTROL_DIM + j] + du_d[CONTROL_DIM*T*(blockDim.x*bdx + tdx) + i*CONTROL_DIM + j]*vars[j];
	}
      }
      //Check to see if the control commands are allowable
      compute_dynamics(s, u, lwpr_input, rfs1, rfs2, rfs3, sigmas, i, numRFS1, numRFS2, numRFS3);
      float inst_cost = compute_cost(s,u,goal_d, sigmas);
      aug_state_costs_d[M*T*((blockDim.x)*bdx + tdx) + T*tdy + i] = inst_cost;
    }
  }	
}

__global__ void expec_costs_kernel(float* state_costs_d, float* aug_state_costs_d)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;
  float expec_cost = 0;
  int i;
  if (tdx < T && bdx < K) {
    for (i = 0; i < M; i++) {
      expec_cost += aug_state_costs_d[M*T*bdx + T*i + tdx];
    }
    state_costs_d[T*bdx + tdx] = expec_cost/(1.0*M);
  }
}

__global__ void norm_exp_costs_kernel(float* state_costs_d)
{
  int tdx = threadIdx.x;
  int bdx = blockIdx.x;
  int index = blockDim.x*bdx + tdx;
  if (index < K) {
    float cost2go = 0;
    float nf_normal = 0;
    int i;
    for (i = T-1; i >= 0; i--) {
      cost2go += state_costs_d[T*index + i];
      nf_normal += state_costs_d[i];
      state_costs_d[T*index + i] = __expf(-10.0*cost2go/nf_normal);
    }
  }
}

//=========================================================================================
//--------------------------------END CUDA------------------------------------------------
//========================================================================================

void compute_control(float* state, float* U, float* goal, LWPR_Model model1, LWPR_Model model2,
		     LWPR_Model model3, float* vars, hiprandGenerator_t gen) {
  
  //Timing Code
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //First we create du_d, perturbations of U which reside in device memory.
  float* du_d;
  HANDLE_ERROR( hipMalloc((void**)&du_d, K*T*CONTROL_DIM*sizeof(float)));
  hiprandGenerateNormal(gen, du_d, K*T*CONTROL_DIM, 0.0, 1.0);
  //Next we create dm_d perturbations of the LWPR model in device memory
  float* dm_temp;
  HANDLE_ERROR( hipMalloc((void**)&dm_temp, M*T*DERIV_STATE_DIM*sizeof(float)));
  hiprandGenerateNormal(gen, dm_temp, M*T*DERIV_STATE_DIM, 0.0, 1.0);
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(dm_d), dm_temp, M*T*DERIV_STATE_DIM*sizeof(float), 0, hipMemcpyDeviceToDevice));
  hipFree(dm_temp);
  //Create pointers for state, U, goal, rfs1, rfs2, and vars in device memory
  float* state_d;
  float* goal_d;
  float* vars_d;
  //Transfer relevant data from host LWPR model to device LWPR Receptive Field
  int i,j;
  RF_Predict* rfs1;
  RF_Predict* rfs2;
  RF_Predict* rfs3;
  rfs1 = (RF_Predict*)malloc(model1.sub[0].numRFS*sizeof(RF_Predict));
  rfs2 = (RF_Predict*)malloc(model2.sub[0].numRFS*sizeof(RF_Predict));
  rfs3 = (RF_Predict*)malloc(model3.sub[0].numRFS*sizeof(RF_Predict)); 
  for (i = 0; i < model1.sub[0].numRFS; i++) {
    rfTransfer(model1.sub[0].rf[i], &rfs1[i], model1.nInStore);
  }
  for (i = 0; i < model2.sub[0].numRFS; i++) {
    rfTransfer(model2.sub[0].rf[i], &rfs2[i], model2.nInStore);
  }
  for (i = 0; i < model3.sub[0].numRFS; i++) {
    rfTransfer(model3.sub[0].rf[i], &rfs3[i], model3.nInStore);
  }
  //Transfer norms to float arrays
  float norm_in[N];
  for (i = 0; i < N; i++) {
    norm_in[i] = float(model1.norm_in[i]);
  }
 //Create device pointers for rfs1, rfs2, norm_in1, and norm_in2
  RF_Predict* rfs1_d;
  RF_Predict* rfs2_d;
  RF_Predict* rfs3_d;
  //Allocate space for state, U, goal, rfs1, rfs2, and vars in device memory
  HANDLE_ERROR( hipMalloc((void**)&state_d, STATE_DIM*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&goal_d, STATE_DIM*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&vars_d, CONTROL_DIM*sizeof(float)));
  HANDLE_ERROR( hipMalloc((void**)&rfs1_d, model1.sub[0].numRFS*sizeof(RF_Predict)));
  HANDLE_ERROR( hipMalloc((void**)&rfs2_d, model2.sub[0].numRFS*sizeof(RF_Predict)));
  HANDLE_ERROR( hipMalloc((void**)&rfs3_d, model3.sub[0].numRFS*sizeof(RF_Predict)));
  //Copy state, U, goal, model1, and model2 into device memory
  HANDLE_ERROR( hipMemcpy(state_d, state, STATE_DIM*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(U_d), U, CONTROL_DIM*T*sizeof(float), 0, hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpy(goal_d, goal, STATE_DIM*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpy(vars_d, vars, CONTROL_DIM*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpy(rfs1_d, rfs1, model1.sub[0].numRFS*sizeof(RF_Predict), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpy(rfs2_d, rfs2, model2.sub[0].numRFS*sizeof(RF_Predict), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpy(rfs3_d, rfs3, model3.sub[0].numRFS*sizeof(RF_Predict), hipMemcpyHostToDevice));
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(norm_in_d), norm_in, N*sizeof(float), 0, hipMemcpyHostToDevice));
  //Allocate space for the state costs and new controls
  //For the raw state costs
  float* aug_state_costs_d;
  HANDLE_ERROR( hipMalloc((void**)&aug_state_costs_d, T*K*M*sizeof(float)));
  //For the averaged state costs
  float* state_costs_d;
  //For controls we just re-use du_d
  HANDLE_ERROR( hipMalloc((void**)&state_costs_d, T*K*sizeof(float)));
  //Now we set the grid and block size
  int xBlockSize = (BLOCKSIZE-1)/M + 1;
  int yBlockSize = M;
  int xGridSize = (K-1)/xBlockSize + 1;
  dim3 dimBlock1(xBlockSize, yBlockSize, 1);
  dim3 dimGrid1(xGridSize, 1, 1);
  hipEventRecord(start, 0);
  //Now we launch the kernel to compute the new control
  rollout_kernel<<<dimGrid1, dimBlock1>>>(aug_state_costs_d, state_d, goal_d, rfs1_d, rfs2_d, rfs3_d, du_d, vars_d, model1.sub[0].numRFS, model2.sub[0].numRFS, model3.sub[0].numRFS);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipDeviceSynchronize();
  //Wait until the kernel has finished
  dim3 dimBlock2(T, 1, 1);
  dim3 dimGrid2(K, 1, 1);
  //Compute expectation of the costs
  expec_costs_kernel<<<dimGrid2, dimBlock2>>>(state_costs_d, aug_state_costs_d);
  hipDeviceSynchronize();
  dim3 dimBlock3(64, 1, 1);
  dim3 dimGrid3((K-1)/64 + 1, 1, 1);
  //Now we normalize the cost-to-go by the noise free path, and exponentiate by the -lambda*cost2go
  norm_exp_costs_kernel<<<dimGrid3, dimBlock3>>>(state_costs_d);
  hipDeviceSynchronize();
  //Compute the normalizer
  //For now just do it on the CPU
  //Transfer state costs to host memory
  float* state_costs;
  state_costs = (float*)malloc(T*K*sizeof(float));
  HANDLE_ERROR( hipMemcpy(state_costs, state_costs_d, T*K*sizeof(float), hipMemcpyDeviceToHost));
  //Now compute the normalizer
  float* normalizer;
  normalizer = (float*)malloc(T*sizeof(float));
  for (i = 0; i < T; i++) {
    normalizer[i] = 0;
    for (j = 0; j < K; j++) {
      normalizer[i] += state_costs[T*j + i];
    }
  }
  //Compute the new controls
  //Just do on CPU for now
  //First transfer controls to host memory
  float* du;
  du = (float*)malloc(T*K*CONTROL_DIM*sizeof(float));
  HANDLE_ERROR( hipMemcpy(du, du_d, T*K*CONTROL_DIM*sizeof(float), hipMemcpyDeviceToHost));
  //Now compute the new control and place it in U
  float* U_new;
  U_new = (float*)malloc(T*CONTROL_DIM*sizeof(float));
  for (i = 0; i < T; i++) {
    U_new[CONTROL_DIM*i] = (state_costs[i]/normalizer[i])*U[CONTROL_DIM*i];
    U_new[CONTROL_DIM*i + 1] = (state_costs[i]/normalizer[i])*U[CONTROL_DIM*i + 1];
    for (j = 0; j < K; j++) {
      float u1 = U[CONTROL_DIM*i] + du[T*CONTROL_DIM*j + CONTROL_DIM*i]*vars[0];
      float u2 = U[CONTROL_DIM*i + 1] + du[T*CONTROL_DIM*j + CONTROL_DIM*i + 1]*vars[1];
      float u_max = .5;
      float u_min = -.5;
      u1 = fmin(u1, u_max);
      u1 = fmax(u1, u_min);
      u2 = fmin(u2, u_max);
      u2 = fmax(u2, u_min);
      U_new[CONTROL_DIM*i] += (state_costs[T*j + i]/normalizer[i])*u1;
      U_new[CONTROL_DIM*i + 1] += (state_costs[T*j + i]/normalizer[i])*u2;
    }
    U[i*CONTROL_DIM] = U_new[i*CONTROL_DIM];
    U[i*CONTROL_DIM + 1] = U_new[i*CONTROL_DIM + 1];
  }
  //Free device arrays
  hipFree(state_d);
  hipFree(goal_d);
  hipFree(rfs1_d);
  hipFree(rfs2_d);
  hipFree(rfs3_d);
  hipFree(du_d);
  hipFree(state_costs_d);
  hipFree(aug_state_costs_d);
  hipFree(vars_d);
  //Free host arrays
  free(rfs1);
  free(rfs2);
  free(rfs3);
  free(state_costs);
  free(du);
  free(normalizer);
  //Print timing results
  hipEventElapsedTime(&time, start, stop); 
  printf("Kernel Time: %f ms \n", time);
}


void dynamics(float* s, float* u, float dt) {
  s[0] += dt*(s[3] + 1.1*s[4])/2.0*cos(s[2]);
  s[1] += dt*(s[3] + 1.1*s[4])/2.0*sin(s[2]);
  s[2] += dt*(s[3] - 1.1*s[4])/.258;
  s[3] += dt*((u[0] + u[1]) - s[3]);
  s[4] += dt*((u[0] - u[1]) - s[4]);
  if (s[0] > 10.0) {
    s[0] = 10.0;
  }
  else if (s[0] < -10.0) {
    s[0] = -10.0;
  }
  if (s[1] > 10.0) {
    s[1] = 10.0;
  }
  else if (s[1] < -10.0) {
    s[1] = -10.0;
  }
  if (s[2] > 3.14) {
    s[2] = -3.14;
  }
  else if (s[2] < -3.14) {
    s[2] = 3.14;
  }
  if (s[3] > .5) {
    s[3] = .5;
  }
  else if (s[3] < -.5) {
    s[3] = -.5;
  }
  if (s[4] > .5) {
    s[4] = .5;
  }
  else if (s[4] < -.5) {
    s[4] = -.5;
  }
}

int main(int argc, char** argv) {
  //Initialize ROS
  ros::init(argc, argv, "pi3_controller");
  ros::NodeHandle n_pub;
  ros::Publisher pi3_pub = n_pub.advertise<pi3_irobot_pkg::rl_vel>("control",1);
  ros::Rate loop_rate(20);
  pi3_irobot_pkg::rl_vel control_msg;
  
  LWPR_Model model1;
  LWPR_Model model2;
  LWPR_Model model3;
  
  char x_dot[] = {'t', 'r', 'a', 'j', '_', 'x', '.', 'x', 'm', 'l', '\0'};
  char y_dot[] = {'t', 'r', 'a', 'j', '_', 'y', '.', 'x', 'm', 'l', '\0'};
  char theta_dot[] = {'t', 'h', 'e', 't', 'a', '.', 'x', 'm', 'l', '\0'};
  int e1[] = {-3};
  int e2[] = {-3};
  int e3[] = {-3};

  lwpr_init_model(&model1, 5, 1, "x");
  lwpr_init_model(&model2, 5, 1, "y");
  lwpr_init_model(&model3, 5, 1, "theta");

  int a1 = lwpr_read_xml(&model1, x_dot, e1);
  int a2 = lwpr_read_xml(&model2, y_dot, e2);
  int a3 = lwpr_read_xml(&model3, theta_dot, e3);
  printf("%d, %d, %d", a1, a2, a3);
  ROS_INFO("%d, %d, %d", e1[0], e2[0], e3[0]);
  
  float U[T*CONTROL_DIM] = {0};
  float u[CONTROL_DIM] = {0};
  
  //Declare a new StateUpdater object
  float s[STATE_DIM] = {0};
  StateUpdater ros_state(s);
  ros_state.init_subscriber();
  
  float goal[] = {1.0, 0, 0, 0, 0};
  float vars[] = {.50, .25};
  
  hiprandGenerator_t gen;
  float dt = (1.0)/(1.0*HZ);
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  
  int i,j,count;
  count = 0;
  while (ros::ok()) {
    count++;
    for (j = 0; j < STATE_DIM; j++) {
      s[j] = ros_state.s[j];
    }
    compute_control(s, U, goal, model1, model2, model3, vars, gen);
    u[0] = U[0];
    u[1] = U[1];
    
    float l_vel = (u[0] + u[1]);
    float r_vel = (u[0] - u[1]);
    
    //Publish the commands
    control_msg.r_vel = r_vel;
    control_msg.l_vel = l_vel;
    pi3_pub.publish(control_msg);
    
    for (i = 0; i < (T-1)*CONTROL_DIM; i++) {
      U[i] = U[i+CONTROL_DIM];
    }
    U[T-2] = 0;
    U[T-1] = 0;

    double lwpr_input[5] = {(double)ros_state.s[0], (double)ros_state.s[1], (double)ros_state.s[2], (double)ros_state.s[3], (double)ros_state.s[4]};
    double out1 = (double)ros_state.s[0];
    double out2 = (double)ros_state.s[1];
    double out3 = (double)ros_state.s[2];
    
    //dynamics(s, u, dt);
    ros::spinOnce();
    
    out1 = (ros_state.s[0] - out1)/dt;
    out2 = (ros_state.s[1] - out2)/dt;
    out3 = (ros_state.s[2] - out3)/dt;
    
    lwpr_update(&model1, lwpr_input, &out1, NULL, NULL);
    lwpr_update(&model2, lwpr_input, &out2, NULL, NULL);
    lwpr_update(&model3, lwpr_input, &out3, NULL, NULL);
    
    printf("Current Location: (%f, %f, %f, %f, %f,) \n", ros_state.s[0], ros_state.s[1], ros_state.s[2], 
	   ros_state.s[3], ros_state.s[4]);
    loop_rate.sleep();
  }
  //Save the LWPR models
  char xn_dot[] = {'x', 'n', '.', 'x', 'm', 'l', '\0'};
  char yn_dot[] = {'y', 'n', '.', 'x', 'm', 'l', '\0'};
  char thetan_dot[] = {'t', 'h', 'e', 't', 'a', 'n', '.', 'x', 'm', 'l', '\0'};
  lwpr_write_xml(&model1, xn_dot);
  lwpr_write_xml(&model2, yn_dot);
  lwpr_write_xml(&model3, thetan_dot);
}
  
